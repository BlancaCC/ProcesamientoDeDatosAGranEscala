// includes, kernels

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#define NUM_ELEMENTS 512
// **===------------------------------------------------------------------===**
//! @param g_idata  input data in global memory
//                  result is expected in index 0 of g_idata
//! @param n        input number of elements to scan from input data
// **===------------------------------------------------------------------===**
__global__ void reduction(float *g_data, const int n)
{
  // params
  int stride_size = ( n - blockDim.x +1) / blockDim.x; // ceil 
  int init = threadIdx.x * stride_size; 
  
  // Define shared memory
  __shared__ float scratch[NUM_ELEMENTS];
  // Load the shared memory
  // Alamcenamos nuestra posición y los contiguos 
  for(int i = init; i < init + stride_size && i < n; i++){
    scratch[i] = g_data[i]; // 
  }
  // Almacenamos los saltos 
  for (int stride = init+stride_size; stride < n; stride+= stride_size)
  {
    scratch[stride] = g_data[stride];
  }

  __syncthreads();
  // Do sum reduction from shared memory
  
  ///////  Reduction scheme 3 //////
  for(int i = init+1; i < init + stride_size && i < n; i++){
    scratch[init] += scratch[i]; // sumamos elementos contiguos 
  }
   __syncthreads(); // agrupamos
  
  // sumamos el resultado del bloque de la derecha
    if (init  + stride_size < n)
        scratch[ init ] +=  scratch[init  + stride_size];
  __syncthreads();
  
  // Store results back to global memory
  if(threadIdx.x == 0)
    g_data[0] = scratch[0];
return; 
}