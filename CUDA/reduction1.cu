// includes, kernels

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#define NUM_ELEMENTS 512
// **===------------------------------------------------------------------===**
//! @param g_idata  input data in global memory
//                  result is expected in index 0 of g_idata
//! @param n        input number of elements to scan from input data
// **===------------------------------------------------------------------===**
__global__ void reduction(float *g_data, int n)
{
  int stride;
  // Define shared memory
  __shared__ float scratch[NUM_ELEMENTS];
  // Load the shared memory
  scratch[threadIdx.x ] = g_data[threadIdx.x];
  if(threadIdx.x + blockDim.x < n)
    scratch[threadIdx.x + blockDim.x] = g_data[threadIdx.x + blockDim.x];
  __syncthreads();
  // Do sum reduction from shared memory
  for(stride = 1 ; stride < blockDim.x; stride *= 2)
  {
    __syncthreads();
    if(threadIdx.x % (2*stride) == 0)
        scratch[threadIdx.x] += scratch[threadIdx.x + stride];
  }
  // Store results back to global memory
  if(threadIdx.x == 0)
    g_data[0] = scratch[0];
return; 
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
void runTest( int argc, char** argv);
float computeOnDevice(float* h_data, int array_mem_size);
extern "C" void computeGold( float* reference, float* idata, const unsigned int len);
int main( int argc, char** argv)
{
    runTest( argc, argv);
    return EXIT_SUCCESS;
}
////////////////////////////////////////////////////////////////////////////////
//! Run naive scan test
////////////////////////////////////////////////////////////////////////////////
void runTest( int argc, char** argv)
{
    int num_elements = NUM_ELEMENTS;
    const unsigned int array_mem_size = sizeof( float) * num_elements;
    // allocate host memory to store the input data
    float* h_data = (float*) malloc( array_mem_size);
    // * No arguments: Randomly generate input data and compare against the host's
            // initialize the input data on the host to be integer values
            // between 0 and 1000
            for( unsigned int i = 0; i < num_elements; ++i)
            {
                //h_data[i] = floorf(1000*(rand()/(float)RAND_MAX));
                h_data[i] = i*1.0;
}
        // compute reference solution
    float reference = 0.0f;
    computeGold(&reference , h_data, num_elements);
    float result = computeOnDevice(h_data, num_elements);
    // We can use an epsilon of 0 since values are integral and in a range
    // that can be exactly represented
    float epsilon = 0.0f;
    unsigned int result_regtest = (abs(result - reference) <= epsilon);
    printf( "Test %s\n", (1 == result_regtest) ? "PASSED" : "FAILED");
    printf( "device: %f  host: %f\n", result, reference);
    // cleanup memory
    free( h_data);
}
/////////////////////////////////////////////////////////////////////////
// Take h_data from host, copies it to device, setup grid and thread
 // dimentions, excutes kernel function, and copy result of scan back
 // to h_data.
 // Note: float* h_data is both the input and the output of this function.
 /////////////////////////////////////////////////////////////////////////
 float computeOnDevice(float* h_data, int num_elements)
 {
   float* d_data = NULL;
   float result;
   // Memory allocation on device side
   hipMalloc((void**)&d_data, num_elements*sizeof(float));
   // Copy from host memory to device memory
   hipMemcpy(d_data, h_data, num_elements*sizeof(float), hipMemcpyHostToDevice);
   //int threads = (num_elements/2) + num_elements%2;
   int threads = num_elements;
   // Invoke the kernel
   reduction<<<1,threads>>>(d_data,num_elements);
   // Copy from device memory back to host memory
   hipMemcpy(&result, d_data, sizeof(float), hipMemcpyDeviceToHost);
   hipFree(d_data);
   return result;
}
 ///////////////////////////////////////////////////////////////////////
 void computeGold( float* reference, float* idata, const unsigned int len)
 {
   reference[0] = 0;
   double total_sum = 0;
   unsigned int i;
   for( i = 0; i < len; ++i)
   {
       total_sum += idata[i];
   }
   reference[0] = total_sum;
 }
 ///////////////////////////////////////////////////////////////////////